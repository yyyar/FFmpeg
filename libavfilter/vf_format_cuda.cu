
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2020 Yaroslav Pogrebnyak <yyyaroslav@gmail.com>
 *
 * This file is part of FFmpeg.
 *
 * FFmpeg is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public
 * License as published by the Free Software Foundation; either
 * version 2.1 of the License, or (at your option) any later version.
 *
 * FFmpeg is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with FFmpeg; if not, write to the Free Software
 * Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA 02110-1301 USA
 */

extern "C" {

__global__ void Format_Cuda_Short_To_Char(
    unsigned short* in_data, int in_linesize,
    unsigned char* out_data, int out_linesize,
    int width, int height, int x_factor)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > width || y > height || x < 0 || y < 0 ) {
        return;
    }

    out_data[x + y*out_linesize ] = in_data[x*x_factor + y*in_linesize ] >> 8;
}

} // extern "C"
